#include <iostream>
#include <hip/hip_runtime.h>

int main()
{
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0); // Get properties of device 0
    std::cout << "Number of SMs: " << prop.multiProcessorCount << std::endl;
    return 0;
}