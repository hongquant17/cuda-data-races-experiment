#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_fp16.h>

#define THREADS_PER_BLOCK 64

__global__ void initCurandStates(hiprandState *states, unsigned long seed)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__device__ __inline__ void artificialDelay(hiprandState *state, int minCycles, int maxCycles)
{
    int cycles = minCycles + (hiprand(state) % (maxCycles - minCycles));
    clock_t start_clock = clock();
    clock_t clock_offset = cycles * 1000;
    while (clock() - start_clock < clock_offset)
        ;
}

template <typename T>
__device__ void printVal(T h_data)
{
    if constexpr (std::is_same<T, float>::value)
    {
        printf("float %f\n", h_data);
    }
    else if constexpr (std::is_same<T, double>::value)
    {
        printf("double %f\n", h_data);
    }
    else if constexpr (std::is_same<T, int8_t>::value)
    {
        printf("int8 %d\n", h_data);
    }
    else if constexpr (std::is_same<T, int16_t>::value)
    {
        printf("int16 %d\n", h_data);
    }
    else if constexpr (std::is_same<T, int32_t>::value)
    {
        printf("int32 %d\n", h_data);
    }
    else if constexpr (std::is_same<T, int64_t>::value)
    {
        printf("int64 %ld\n", h_data);
    }
    else if constexpr (std::is_same<T, u_int8_t>::value)
    {
        printf("u_int8 %d\n", h_data);
    }
    else if constexpr (std::is_same<T, u_int16_t>::value)
    {
        printf("u_int16 %d\n", h_data);
    }
    else if constexpr (std::is_same<T, u_int32_t>::value)
    {
        printf("u_int32 %d\n", h_data);
    }
    else if constexpr (std::is_same<T, u_int64_t>::value)
    {
        printf("u_int64 %ld\n", h_data);
    }
    else if constexpr (std::is_same<T, __half>::value)
    {
        printf("__half %f\n\n", __half2float(h_data));
    }
}

__device__ int getSMID()
{
    int smid;
    asm("mov.u32 %0, %smid;" : "=r"(smid));
    return smid;
}

template <typename T>
__global__ void dataRaceKernelSameWarp(T *data, T *buffer, hiprandState *randStates, int thread1, int thread2)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int smid = getSMID();
    // if (gridDim.x > 1 and (blockIdx.x == 0 or blockIdx.x == gridDim.x - 1)) {
    //     printf("Block [%d] is on SM %d\n", blockIdx.x, smid);
    // }
    if (idx == thread1)
    {
        // artificialDelay(&randStates[idx], 100, 200);
        data[0] = idx;
    }
    else if (idx == thread2)
    {
        // artificialDelay(&randStates[idx], 100, 200);
        buffer[0] = data[0];
    }
}

template <typename T>
void experiment1()
{
    T *d_data, *d_buffer;
    T h_data;
    hiprandState *d_randStates;

    hipMalloc(&d_data, sizeof(T));
    hipMalloc(&d_buffer, sizeof(T));
    hipMalloc(&d_randStates, THREADS_PER_BLOCK * sizeof(hiprandState));

    h_data = static_cast<T>(-1);
    hipMemcpy(d_data, &h_data, sizeof(T), hipMemcpyHostToDevice);

    initCurandStates<<<1, THREADS_PER_BLOCK>>>(d_randStates, 0);
    hipDeviceSynchronize();

    printf("Experiment 1 (Same Warp): ");
    dataRaceKernelSameWarp<<<1, THREADS_PER_BLOCK>>>(d_data, d_buffer, d_randStates, 2, 3);
    hipDeviceSynchronize();

    hipMemcpy(&h_data, d_buffer, sizeof(T), hipMemcpyDeviceToHost);
    printVal(h_data);

    hipFree(d_data);
    hipFree(d_randStates);
    hipFree(d_buffer);
}

template <typename T>
void experiment2()
{
    T *d_data, *d_buffer;
    T h_data;
    hiprandState *d_randStates;

    hipMalloc(&d_data, sizeof(T));
    hipMalloc(&d_buffer, sizeof(T));
    hipMalloc(&d_randStates, THREADS_PER_BLOCK * sizeof(hiprandState));

    h_data = static_cast<T>(-1);
    hipMemcpy(d_data, &h_data, sizeof(T), hipMemcpyHostToDevice);

    initCurandStates<<<1, THREADS_PER_BLOCK>>>(d_randStates, 0);
    hipDeviceSynchronize();

    printf("Experiment 2 (Different Warp Same Block): ");
    dataRaceKernelSameWarp<<<1, THREADS_PER_BLOCK>>>(d_data, d_buffer, d_randStates, 2, 50);
    hipDeviceSynchronize();

    hipMemcpy(&h_data, d_buffer, sizeof(T), hipMemcpyDeviceToHost);
    printVal(h_data);

    hipFree(d_data);
    hipFree(d_randStates);
    hipFree(d_buffer);
}

template <typename T>
void experiment3()
{
    T *d_data, *d_buffer;
    T h_data;
    hiprandState *d_randStates;

    hipMalloc(&d_data, sizeof(T));
    hipMalloc(&d_buffer, sizeof(T));
    hipMalloc(&d_randStates, THREADS_PER_BLOCK * sizeof(hiprandState));

    h_data = static_cast<T>(-1);
    hipMemcpy(d_data, &h_data, sizeof(T), hipMemcpyHostToDevice);

    initCurandStates<<<21, THREADS_PER_BLOCK>>>(d_randStates, 0);
    hipDeviceSynchronize();

    printf("Experiment 3 (Different Block Same SM): ");
    dataRaceKernelSameWarp<<<21, THREADS_PER_BLOCK>>>(d_data, d_buffer, d_randStates, 2, 1280);
    hipDeviceSynchronize();

    hipMemcpy(&h_data, d_buffer, sizeof(T), hipMemcpyDeviceToHost);
    printVal(h_data);

    hipFree(d_data);
    hipFree(d_randStates);
    hipFree(d_buffer);
}

template <typename T>
void experiment4()
{
    T *d_data, *d_buffer;
    T h_data;
    hiprandState *d_randStates;

    hipMalloc(&d_data, sizeof(T));
    hipMalloc(&d_buffer, sizeof(T));
    hipMalloc(&d_randStates, THREADS_PER_BLOCK * sizeof(hiprandState));

    h_data = static_cast<T>(-1);
    hipMemcpy(d_data, &h_data, sizeof(T), hipMemcpyHostToDevice);

    initCurandStates<<<16, THREADS_PER_BLOCK>>>(d_randStates, 0);
    hipDeviceSynchronize();

    printf("Experiment 4 (Different SM): ");

    dataRaceKernelSameWarp<<<16, THREADS_PER_BLOCK>>>(d_data, d_buffer, d_randStates, 2, 70);
    hipDeviceSynchronize();

    hipMemcpy(&h_data, d_buffer, sizeof(T), hipMemcpyDeviceToHost);
    printVal(h_data);

    hipFree(d_data);
    hipFree(d_randStates);
    hipFree(d_buffer);
}

int main(int argc, char **argv)
{
    printf("Running experiments to observe data races:\n\n");

    {
        experiment1<float>();
        hipDeviceSynchronize();

        experiment1<double>();
        hipDeviceSynchronize();

        experiment1<int8_t>();
        hipDeviceSynchronize();

        experiment1<int16_t>();
        hipDeviceSynchronize();

        experiment1<int32_t>();
        hipDeviceSynchronize();

        experiment1<int64_t>();
        hipDeviceSynchronize();

        experiment1<u_int8_t>();
        hipDeviceSynchronize();

        experiment1<u_int16_t>();
        hipDeviceSynchronize();

        experiment1<u_int32_t>();
        hipDeviceSynchronize();

        experiment1<u_int64_t>();
        hipDeviceSynchronize();

        experiment1<__half>();
        hipDeviceSynchronize();
    }

    {
        experiment2<float>();
        hipDeviceSynchronize();

        experiment2<double>();
        hipDeviceSynchronize();

        experiment2<int8_t>();
        hipDeviceSynchronize();

        experiment2<int16_t>();
        hipDeviceSynchronize();

        experiment2<int32_t>();
        hipDeviceSynchronize();

        experiment2<int64_t>();
        hipDeviceSynchronize();

        experiment2<u_int8_t>();
        hipDeviceSynchronize();

        experiment2<u_int16_t>();
        hipDeviceSynchronize();

        experiment2<u_int32_t>();
        hipDeviceSynchronize();

        experiment2<u_int64_t>();
        hipDeviceSynchronize();

        experiment2<__half>();
        hipDeviceSynchronize();
    }

    {
        experiment3<float>();
        hipDeviceSynchronize();

        experiment3<double>();
        hipDeviceSynchronize();

        experiment3<int8_t>();
        hipDeviceSynchronize();

        experiment3<int16_t>();
        hipDeviceSynchronize();

        experiment3<int32_t>();
        hipDeviceSynchronize();

        experiment3<int64_t>();
        hipDeviceSynchronize();

        experiment3<u_int8_t>();
        hipDeviceSynchronize();

        experiment3<u_int16_t>();
        hipDeviceSynchronize();

        experiment3<u_int32_t>();
        hipDeviceSynchronize();

        experiment3<u_int64_t>();
        hipDeviceSynchronize();

        experiment3<__half>();
        hipDeviceSynchronize();
    }

    {
        experiment4<float>();
        hipDeviceSynchronize();

        experiment4<double>();
        hipDeviceSynchronize();

        experiment4<int8_t>();
        hipDeviceSynchronize();

        experiment4<int16_t>();
        hipDeviceSynchronize();

        experiment4<int32_t>();
        hipDeviceSynchronize();

        experiment4<int64_t>();
        hipDeviceSynchronize();

        experiment4<u_int8_t>();
        hipDeviceSynchronize();

        experiment4<u_int16_t>();
        hipDeviceSynchronize();

        experiment4<u_int32_t>();
        hipDeviceSynchronize();

        experiment4<u_int64_t>();
        hipDeviceSynchronize();

        experiment4<__half>();
        hipDeviceSynchronize();
    }

    return 0;
}
