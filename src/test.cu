#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>

#define THREADS_PER_BLOCK 1024

__global__ void initCurandStates(hiprandState *states, unsigned long seed)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__device__ __inline__ void artificialDelay(hiprandState *state, int minCycles, int maxCycles)
{
    int cycles = minCycles + (hiprand(state) % (maxCycles - minCycles));
    clock_t start_clock = clock();
    clock_t clock_offset = cycles * 1000;
    while (clock() - start_clock < clock_offset) ;
}

__global__ void dataRaceKernelSameWarp(int *data, hiprandState *randStates)
{
    int idx = threadIdx.x;
    if (idx < 2)
    {
        artificialDelay(&randStates[idx], 100, 200);
        data[0] = idx;
    }
}

void experiment1()
{
    int *d_data;
    int h_data;

    hipMalloc((void **)&d_data, sizeof(int));
    h_data = -1;
    hipMemcpy(d_data, &h_data, sizeof(int), hipMemcpyHostToDevice);

    hiprandState *d_randStates;
    hipMalloc((void **)&d_randStates, THREADS_PER_BLOCK * sizeof(hiprandState));
    initCurandStates<<<1, THREADS_PER_BLOCK>>>(d_randStates, time(0));
    hipDeviceSynchronize();

    dataRaceKernelSameWarp<<<1, THREADS_PER_BLOCK>>>(d_data, d_randStates);
    hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost);
    printf("Experiment 1 (Same Warp): %d\n", h_data);

    hipFree(d_data);
    hipFree(d_randStates);
}

__global__ void dataRaceKernelDifferentWarpSameBlock(int *data, hiprandState *randStates)
{
    int idx = threadIdx.x;

    if (idx == 1 || idx == 1000)
    {
        artificialDelay(&randStates[idx], 100, 200);
        data[0] = idx;
    }
}

void experiment2()
{
    int *d_data;
    int h_data;

    hipMalloc((void **)&d_data, sizeof(int));
    h_data = -1;
    hipMemcpy(d_data, &h_data, sizeof(int), hipMemcpyHostToDevice);

    hiprandState *d_randStates;
    hipMalloc((void **)&d_randStates, THREADS_PER_BLOCK * sizeof(hiprandState));
    initCurandStates<<<1, THREADS_PER_BLOCK>>>(d_randStates, time(0));
    hipDeviceSynchronize();

    dataRaceKernelDifferentWarpSameBlock<<<1, THREADS_PER_BLOCK>>>(d_data, d_randStates);
    hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost);
    printf("Experiment 2 (Different Warp Same Block): %d\n", h_data);

    hipFree(d_data);
    hipFree(d_randStates);
}

__global__ void dataRaceKernelDifferentBlocksSameSM(int *data, hiprandState *randStates)
{
    int idx = threadIdx.x;

    if (idx == 0)
    {
        artificialDelay(&randStates[idx], 100, 200);
        data[0] = blockIdx.x;
    }
}

void experiment3()
{
    int *d_data;
    int h_data;

    hipMalloc((void **)&d_data, sizeof(int));
    h_data = -1;
    hipMemcpy(d_data, &h_data, sizeof(int), hipMemcpyHostToDevice);

    hiprandState *d_randStates;
    hipMalloc((void **)&d_randStates, THREADS_PER_BLOCK * sizeof(hiprandState));
    initCurandStates<<<2, THREADS_PER_BLOCK>>>(d_randStates, time(0));
    hipDeviceSynchronize();

    dataRaceKernelDifferentBlocksSameSM<<<2, THREADS_PER_BLOCK>>>(d_data, d_randStates);
    hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost);
    printf("Experiment 3 (Different Blocks Same SM): %d\n", h_data);

    hipFree(d_data);
    hipFree(d_randStates);
}

__global__ void dataRaceKernelDifferentSMs(int *data, hiprandState *randStates)
{
    int idx = threadIdx.x;

    if (idx == 0 && (blockIdx.x == 0 || blockIdx.x == (gridDim.x - 1)))
    {
        artificialDelay(&randStates[idx], 100, 200);
        data[0] = blockIdx.x;
    }
}

void experiment4()
{
    int *d_data;
    int h_data;

    hipMalloc((void **)&d_data, sizeof(int));
    h_data = -1;
    hipMemcpy(d_data, &h_data, sizeof(int), hipMemcpyHostToDevice);

    hiprandState *d_randStates;
    hipMalloc((void **)&d_randStates, THREADS_PER_BLOCK * sizeof(hiprandState));
    initCurandStates<<<12, THREADS_PER_BLOCK>>>(d_randStates, time(0));
    hipDeviceSynchronize();

    int numBlocks = 12;

    dataRaceKernelDifferentSMs<<<numBlocks, THREADS_PER_BLOCK>>>(d_data, d_randStates);
    hipMemcpy(&h_data, d_data, sizeof(int), hipMemcpyDeviceToHost);
    printf("Experiment 4 (Different SMs): %d\n", h_data);

    hipFree(d_data);
    hipFree(d_randStates);
}

int main(int argc, char **argv)
{
    printf("Running experiments to observe data races:\n\n");

    // experiment1();
    // hipDeviceSynchronize();

    // experiment2();
    // hipDeviceSynchronize();

    // experiment3();
    // hipDeviceSynchronize();

    experiment4();
    hipDeviceSynchronize();

    return 0;
}
