#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_fp16.h>

#define THREADS_PER_BLOCK 1024

__global__ void initCurandStates(hiprandState *states, unsigned long seed)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    hiprand_init(seed, idx, 0, &states[idx]);
}

__device__ __inline__ void artificialDelay(hiprandState *state, int minCycles, int maxCycles)
{
    int cycles = minCycles + (hiprand(state) % (maxCycles - minCycles));
    clock_t start_clock = clock();
    clock_t clock_offset = cycles * 1000;
    while (clock() - start_clock < clock_offset)
        ;
}

template <typename T>
void printVal(T h_data) {
    if constexpr (std::is_same<T, float>::value)
    {
        printf("float %f\n", h_data);
    }
    else if constexpr(std::is_same<T, double>::value) {
        printf("double %f\n", h_data);
    }
    else if constexpr(std::is_same<T, int8_t>::value) {
        printf("int8 %d\n", h_data);
    }
    else if constexpr (std::is_same<T, int16_t>::value)
    {
        printf("int16 %d\n", h_data);
    }
    else if constexpr (std::is_same<T, int32_t>::value)
    {
        printf("int32 %d\n", h_data);
    }
    else if constexpr (std::is_same<T, int64_t>::value)
    {
        printf("int64 %ld\n", h_data);
    }
    else if constexpr(std::is_same<T, u_int8_t>::value) {
        printf("u_int8 %d\n", h_data);
    }
    else if constexpr(std::is_same<T, u_int16_t>::value) {
        printf("u_int16 %d\n", h_data);
    }
    else if constexpr(std::is_same<T, u_int32_t>::value) {
        printf("u_int32 %d\n", h_data);
    }
    else if constexpr(std::is_same<T, u_int64_t>::value) {
        printf("u_int64 %ld\n", h_data);
    }
    else if constexpr(std::is_same<T, __half>::value) {
        printf("__half %f\n\n", __half2float(h_data));
    }
}

template <typename T>
__global__ void dataRaceKernelSameWarp(T *data, hiprandState *randStates)
{
    int idx = threadIdx.x;
    if (idx < 3 && idx > 0)
    {
        // artificialDelay(&randStates[idx], 100, 200);
        data[0] = idx;
    }
}

template <typename T>
void experiment1()
{
    T *d_data;
    T h_data;

    hipMalloc((void **)&d_data, sizeof(T));
    h_data = static_cast<T>(-1);
    hipMemcpy(d_data, &h_data, sizeof(T), hipMemcpyHostToDevice);

    hiprandState *d_randStates;
    hipMalloc((void **)&d_randStates, THREADS_PER_BLOCK * sizeof(hiprandState));
    // initCurandStates<<<1, THREADS_PER_BLOCK>>>(d_randStates, time(0));
    // hipDeviceSynchronize();

    dataRaceKernelSameWarp<<<1, THREADS_PER_BLOCK>>>(d_data, d_randStates);
    hipMemcpy(&h_data, d_data, sizeof(T), hipMemcpyDeviceToHost);


    printf("Experiment 1 (Same Warp): ");
    printVal(h_data);

    hipFree(d_data);
    hipFree(d_randStates);
}

template <typename T>
__global__ void dataRaceKernelDifferentWarpSameBlock(T *data, hiprandState *randStates)
{
    int idx = threadIdx.x;

    if (idx == 2 || idx == 60)
    {
        // artificialDelay(&randStates[idx], 100, 200);
        data[0] = idx;
    }
}

template <typename T>
void experiment2()
{
    T *d_data;
    T h_data;

    hipMalloc((void **)&d_data, sizeof(T));
    h_data = static_cast<T>(-1);
    hipMemcpy(d_data, &h_data, sizeof(T), hipMemcpyHostToDevice);

    hiprandState *d_randStates;
    hipMalloc((void **)&d_randStates, THREADS_PER_BLOCK * sizeof(hiprandState));
    // initCurandStates<<<1, THREADS_PER_BLOCK>>>(d_randStates, time(0));
    // hipDeviceSynchronize();

    dataRaceKernelDifferentWarpSameBlock<<<1, THREADS_PER_BLOCK>>>(d_data, d_randStates);
    hipMemcpy(&h_data, d_data, sizeof(T), hipMemcpyDeviceToHost);
    printf("Experiment 2 (Different Warp Same Block): ");
    printVal(h_data);

    hipFree(d_data);
    hipFree(d_randStates);
}

template <typename T>
__global__ void dataRaceKernelDifferentBlocksSameSM(T *data, hiprandState *randStates)
{
    int idx = threadIdx.x;

    if (idx == 0)
    {
        artificialDelay(&randStates[idx], 100, 200);
        data[0] = blockIdx.x;
    }
}

template <typename T>
void experiment3()
{
    T *d_data;
    T h_data;

    hipMalloc((void **)&d_data, sizeof(T));
    h_data = static_cast<T>(-1);
    hipMemcpy(d_data, &h_data, sizeof(T), hipMemcpyHostToDevice);

    hiprandState *d_randStates;
    hipMalloc((void **)&d_randStates, THREADS_PER_BLOCK * sizeof(hiprandState));
    initCurandStates<<<2, THREADS_PER_BLOCK>>>(d_randStates, time(0));
    hipDeviceSynchronize();

    dataRaceKernelDifferentBlocksSameSM<<<2, THREADS_PER_BLOCK>>>(d_data, d_randStates);
    hipMemcpy(&h_data, d_data, sizeof(T), hipMemcpyDeviceToHost);
    printf("Experiment 3 (Different Blocks Same SM): ");
    printVal(h_data);

    hipFree(d_data);
    hipFree(d_randStates);
}

__device__ int getSMID()
{
    int smid;
    asm("mov.u32 %0, %smid;" : "=r"(smid));
    return smid;
}

template <typename T>
__global__ void dataRaceKernelDifferentSMs(T *data, hiprandState *randStates)
{
    int idx = threadIdx.x;
    // int smid = getSMID();
    if (idx == 0 && (blockIdx.x == 0 || blockIdx.x == (gridDim.x - 1)))
    {
        // printf("Block [%d] is on SM %d\n", blockIdx.x, smid);
        // artificialDelay(&randStates[idx], 100, 200);
        data[0] = blockIdx.x;
    }
}

template <typename T>
void experiment4()
{
    T *d_data;
    T h_data;

    hipMalloc((void **)&d_data, sizeof(T));
    h_data = static_cast<T>(-1);
    hipMemcpy(d_data, &h_data, sizeof(T), hipMemcpyHostToDevice);

    int numBlocks = 12;

    hiprandState *d_randStates;
    hipMalloc((void **)&d_randStates, THREADS_PER_BLOCK * sizeof(hiprandState));
    // initCurandStates<<<numBlocks, THREADS_PER_BLOCK>>>(d_randStates, time(0));
    // hipDeviceSynchronize();


    dataRaceKernelDifferentSMs<<<numBlocks, THREADS_PER_BLOCK>>>(d_data, d_randStates);
    hipMemcpy(&h_data, d_data, sizeof(T), hipMemcpyDeviceToHost);
    printf("Experiment 4 (Different SMs): ");
    printVal(h_data);

    hipFree(d_data);
    hipFree(d_randStates);
}

int main(int argc, char **argv)
{
    printf("Running experiments to observe data races:\n\n");


    {
        experiment1<float>();
        hipDeviceSynchronize();

        experiment1<double>();
        hipDeviceSynchronize();

        experiment1<int8_t>();
        hipDeviceSynchronize();

        experiment1<int16_t>();
        hipDeviceSynchronize();

        experiment1<int32_t>();
        hipDeviceSynchronize();

        experiment1<int64_t>();
        hipDeviceSynchronize();

        experiment1<u_int8_t>();
        hipDeviceSynchronize();

        experiment1<u_int16_t>();
        hipDeviceSynchronize();

        experiment1<u_int32_t>();
        hipDeviceSynchronize();

        experiment1<u_int64_t>();
        hipDeviceSynchronize();

        experiment1<__half>();
        hipDeviceSynchronize();
    }

    {
        experiment2<float>();
        hipDeviceSynchronize();

        experiment2<double>();
        hipDeviceSynchronize();

        experiment2<int8_t>();
        hipDeviceSynchronize();

        experiment2<int16_t>();
        hipDeviceSynchronize();

        experiment2<int32_t>();
        hipDeviceSynchronize();

        experiment2<int64_t>();
        hipDeviceSynchronize();

        experiment2<u_int8_t>();
        hipDeviceSynchronize();

        experiment2<u_int16_t>();
        hipDeviceSynchronize();

        experiment2<u_int32_t>();
        hipDeviceSynchronize();

        experiment2<u_int64_t>();
        hipDeviceSynchronize();

        experiment2<__half>();
        hipDeviceSynchronize();
    }

    {
        // experiment3<float>();
        // hipDeviceSynchronize();

        // experiment3<double>();
        // hipDeviceSynchronize();

        // experiment3<int8_t>();
        // hipDeviceSynchronize();

        // experiment3<int16_t>();
        // hipDeviceSynchronize();

        // experiment3<int32_t>();
        // hipDeviceSynchronize();

        // experiment3<int64_t>();
        // hipDeviceSynchronize();

        // experiment3<u_int8_t>();
        // hipDeviceSynchronize();

        // experiment3<u_int16_t>();
        // hipDeviceSynchronize();

        // experiment3<u_int32_t>();
        // hipDeviceSynchronize();

        // experiment3<u_int64_t>();
        // hipDeviceSynchronize();

        // experiment3<__half>();
        // hipDeviceSynchronize();
    }

    {
        experiment4<float>();
        hipDeviceSynchronize();

        experiment4<double>();
        hipDeviceSynchronize();

        experiment4<int8_t>();
        hipDeviceSynchronize();

        experiment4<int16_t>();
        hipDeviceSynchronize();

        experiment4<int32_t>();
        hipDeviceSynchronize();

        experiment4<int64_t>();
        hipDeviceSynchronize();

        experiment4<u_int8_t>();
        hipDeviceSynchronize();

        experiment4<u_int16_t>();
        hipDeviceSynchronize();

        experiment4<u_int32_t>();
        hipDeviceSynchronize();

        experiment4<u_int64_t>();
        hipDeviceSynchronize();

        experiment4<__half>();
        hipDeviceSynchronize();
    }

    return 0;
}
